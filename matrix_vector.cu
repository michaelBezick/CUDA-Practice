#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Kernel: you need to implement this
__global__ void matvec_kernel(const float *A, const float *x, float *y,
                              int rows, int cols) {
  int row = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < rows) {
    float sum = 0;
    for (int i = 0; i < cols; i++) {
      sum += A[row * cols + i] * x[i];
    }

    y[row] = sum;
  }
}

int main() {
  int rows = 4;
  int cols = 3;

  size_t size_A = rows * cols * sizeof(float);
  size_t size_x = cols * sizeof(float);
  size_t size_y = rows * sizeof(float);

  // Host allocations
  float *h_A = (float *)malloc(size_A);
  float *h_x = (float *)malloc(size_x);
  float *h_y = (float *)malloc(size_y);

  // Initialize A and x with simple values
  for (int i = 0; i < rows * cols; i++) {
    h_A[i] = 1.0f; // all ones
  }
  for (int j = 0; j < cols; j++) {
    h_x[j] = (float)(j + 1); // 1, 2, 3, ...
  }

  // Device allocations
  float *d_A, *d_x, *d_y;
  hipMalloc((void **)&d_A, size_A);
  hipMalloc((void **)&d_x, size_x);
  hipMalloc((void **)&d_y, size_y);

  // Copy inputs
  hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
  hipMemcpy(d_x, h_x, size_x, hipMemcpyHostToDevice);

  // Launch kernel
  int threadsPerBlock = 128;
  int blocks = (rows + threadsPerBlock - 1) / threadsPerBlock;
  matvec_kernel<<<blocks, threadsPerBlock>>>(d_A, d_x, d_y, rows, cols);

  // Copy result back
  hipMemcpy(h_y, d_y, size_y, hipMemcpyDeviceToHost);

  // Print result
  printf("Result y:\n");
  for (int i = 0; i < rows; i++) {
    printf("%f\n", h_y[i]);
  }

  // Cleanup
  hipFree(d_A);
  hipFree(d_x);
  hipFree(d_y);
  free(h_A);
  free(h_x);
  free(h_y);

  return 0;
}
